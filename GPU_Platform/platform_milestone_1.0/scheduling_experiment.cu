#include "hip/hip_runtime.h"
#include "scheduling_experiment.h"

void cpuTaskFunc(float cpuTaskLen) {
  float c = 0;
  long i = cpuTaskLen * CPU_UNIT_TASK;
  for (long j = 0; j < i; ++j) {
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
  }
}

__global__ void gpuTaskFunc(int _tid, float gpuTaskLen) {
  long i = threadIdx.x + blockIdx.x * blockDim.x;
  long j = gpuTaskLen * GPU_UNIT_TASK;
  float a = 9876.54321, b = 543.21;
  for (long k = 0; k < j; ++k) {
    deviceData[_tid][i] += a / b;
    deviceData[_tid][i] -= a / b;
  }
}

void *threadFunc(void *_tidPtr) {
  int _tid = *(int *)_tidPtr;
  // convert ddl to microsecond, consistent with duration
  long ddlusec = ddls[_tid] * 1000;

  // pin to either core 4 or 5
  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(4, &cpuSet);
  CPU_SET(5, &cpuSet);
  pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet);

  // set scheduling config
  struct sched_param schedParam;
  schedParam.sched_priority = prios[_tid];
  pthread_setschedparam(pthread_self(), SCHED_FIFO, &schedParam);

  struct timeval startTime;
  struct timeval endTime;
  long duration; // microsecond

  // MAIN LOOP
  for (int i = 0; i < 100; ++i) {
    // launch a batch of tasks
    gettimeofday(&startTime, NULL);
    for (int j = 0; j < gpuTaskNum; ++j) {
      cpuTaskFunc(cpuTaskLens[_tid][j]);
      gpuTaskFunc<<<2, 1024, 0, streams[_tid]>>>(_tid, gpuTaskLens[_tid][j]);
      usleep(gpuTaskLens[_tid][j] * 1000);
      hipStreamSynchronize(streams[_tid]);
    }
    cpuTaskFunc(cpuTaskLens[_tid][cpuTaskNum - 1]);
    gettimeofday(&endTime, NULL);

    // some other pthreads time exceeded
    if (timeExceeded) {
      ++doneCount;
      return NULL;
    }
    // current pthread time exceeded
    duration = endTime.tv_sec * 1000000 + endTime.tv_usec - (startTime.tv_sec * 1000000 + startTime.tv_usec);
    if (duration > ddlusec) {
      timeExceeded = 1;
      ++doneCount;
      return NULL;
    }
    // current pthread not time exceeded, sleep until deadline
    usleep(ddlusec - duration);
  }
  // current pthread successfully schedule
  ++doneCount;
  return NULL;
}

void *timerFunc(void *) {
  sleep(180);
  cout << "Unexpected blocking, abort!" << endl;
  exit(-1);
  return NULL;
}

void prioGen(vector<vector<int>> &rst, int n) {
  int row = 0;
  vector<int> c(n, 0);
  vector<int> A = {95, 96, 97, 98, 99};
  rst[row] = A;
  ++row;
  int i = 0;
  while (i < n) {
    if (c[i] < i) {
      if (i / 2 * 2 == i) {
        swap(A[0], A[i]);
      } else {
        swap(A[c[i]], A[i]);
      }
      rst[row] = A;
      ++row;
      ++c[i];
      i = 0;
    } else {
      c[i] = 0;
      ++i;
    }
  }
}

int main(int argc, char **argv) {
  // init setup
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    hipStreamCreate(&streams[_tid]);
  }
  srand((unsigned)time(NULL));
  // cpu sleep when synchronizing
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

  // read parameters from pthreadData.dat
  ifstream pthreadData;
  pthreadData.open("pthreadData.dat");
  pthreadData >> cpuTaskNum >> gpuTaskNum;
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    pthreadData >> utilRates[_tid] >> ddls[_tid];
    for (int i = 0; i < cpuTaskNum; ++i) {
      pthreadData >> cpuTaskLens[_tid][i];
    }
    for (int i = 0; i < gpuTaskNum; ++i) {
      pthreadData >> gpuTaskLens[_tid][i];
    }
  }
  pthreadData.close();

  // print info
  // for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
  //   cout << "util " << _tid << ": " << utilRates[_tid] << endl << "ddl " << _tid << ": " << ddls[_tid] << endl;
  //   for (int i = 0; i < cpuTaskNum; ++i) {
  //     cout << cpuTaskLens[_tid][i] << " ";
  //   }
  //   cout << '\n';
  //   for (int i = 0; i < gpuTaskNum; ++i) {
  //     cout << gpuTaskLens[_tid][i] << " ";
  //   }
  //   cout << '\n' << endl;
  // }

  // set priorities
  vector<vector<int>> prioPermu(120);
  prioGen(prioPermu, PTHREAD_NUM);
  int nthPermu = atoi(argv[1]);
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    prios[_tid] = prioPermu[nthPermu][_tid];
  }

  // warm up gpu, should be removed in later versions
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    gpuTaskFunc<<<2, 1024, 0, streams[_tid]>>>(_tid, 1000);
  }
  hipDeviceSynchronize();

  // create pthreads
  int _tids[PTHREAD_NUM];
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    _tids[_tid] = _tid;
    pthread_create(&pthreads[_tid], NULL, threadFunc, (void *)&_tids[_tid]);
  }
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    pthread_join(pthreads[_tid], NULL);
  }
  hipDeviceReset();
  return timeExceeded;
}
