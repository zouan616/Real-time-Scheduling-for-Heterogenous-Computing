#include "hip/hip_runtime.h"
// sudo ./scheduling_experiment [priorityOrder]

#include "scheduling_experiment.h"

int fdMinGpuFreq;
int fdMaxGpuFreq;
set<float> gpuRatioTree;
unsigned long gpuAvaiFreqs[] = {
    114750000, 216750000, 318750000,  420750000,  522750000,  624750000, 726750000,
    854250000, 930750000, 1032750000, 1122000000, 1236750000, 1300500000}; // default = 114750000
const char * gpuAvaiFreqs_cpy[] = {
    "114750000", "216750000", "318750000",  "420750000",  "522750000",  "624750000", "726750000",
    "854250000", "930750000", "1032750000", "1122000000", "1236750000", "1300500000"};

void cpuTaskFunc(float cpuTaskLen) {
  float c = 0;
  long i = cpuTaskLen * CPU_UNIT_TASK;
  for (long j = 0; j < i; ++j) {
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
  }
}

__global__ void gpuTaskFunc(int _tid, float gpuTaskLen) {
  long i = threadIdx.x + blockIdx.x * blockDim.x;
  long j = gpuTaskLen * GPU_UNIT_TASK;
  float a = 9876.54321, b = 543.21;
  for (long k = 0; k < j; ++k) {
    deviceData[_tid][i] += a / b;
    deviceData[_tid][i] -= a / b;
  }
}

void *threadFunc(void *_tidPtr) {
  int _tid = *(int *)_tidPtr;
  long ddlusec = ddls[_tid] * 1000;

  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(1, &cpuSet);
  //CPU_SET(7, &cpuSet);
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  struct sched_param schedParam;
  schedParam.sched_priority = prios[_tid];
  debugCall(pthread_setschedparam(pthread_self(), SCHED_FIFO, &schedParam));

  struct timeval startTime;
  struct timeval endTime;
  long duration; // us

  struct timeval cpuSegStartTime;
  struct timeval cpuSegEndTime;
  long cpuSegDuration; // us
  long diffDuration;   // us
  float gpuRatio;
  float targetGpu;

  // MAIN LOOP
  for (int i = 0; i < 20; ++i) {
    gettimeofday(&startTime, NULL);
    for (int j = 0; j < cpuTaskNum[_tid] - 1; ++j) {
      gettimeofday(&cpuSegStartTime, NULL);
      cpuTaskFunc(cpuTaskLens[_tid][j]);
      gettimeofday(&cpuSegEndTime, NULL);
      cpuSegDuration = cpuSegEndTime.tv_sec * 1000000 + cpuSegEndTime.tv_usec -
                       (cpuSegStartTime.tv_sec * 1000000 + cpuSegStartTime.tv_usec);
      diffDuration = cpuTaskLens[_tid][j] * 1000 - cpuSegDuration;
      if (diffDuration > 0) {
        gpuRatio = (diffDuration + gpuTaskLens[_tid][j] * 1000) / (gpuTaskLens[_tid][j] * 1000);
	targetGpu = 1300500000 / gpuRatio;
	// printf("targetGpu = %f\n", targetGpu);
	for (int k = 0; k < 13; k++){
	  if (gpuAvaiFreqs[k] >= targetGpu){
	    ddlusec += 2000;
	    syscall(SYS_write, fdMaxGpuFreq, gpuAvaiFreqs_cpy[k], 10);
	    syscall(SYS_write, fdMinGpuFreq, gpuAvaiFreqs_cpy[k], 10);
	    break;
	  }	
	}
      }
      gpuTaskFunc<<<1, 2048, 0, cudaStreams[_tid]>>>(_tid, gpuTaskLens[_tid][j]);
    }
    cpuTaskFunc(cpuTaskLens[_tid][cpuTaskNum[_tid] - 1]);
    gettimeofday(&endTime, NULL);

    // some other pthreads time exceeded
    if (timeExceeded) {
      return NULL;
    }
    // current pthread time exceeded
    duration = endTime.tv_sec * 1000000 + endTime.tv_usec - (startTime.tv_sec * 1000000 + startTime.tv_usec);
    if (duration > ddlusec) {
      timeExceeded = 1;
      return NULL;
    }
    // current pthread not time exceeded, sleep until deadline
    usleep(ddlusec - duration);
  }
  // current pthread successfully scheduled
  return NULL;
}

void *syncFunc(void *_tidPtr) {
  int _tid = *(int *)_tidPtr;

  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(0, &cpuSet);
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  struct sched_param schedParam;
  schedParam.sched_priority = 99;
  debugCall(pthread_setschedparam(pthread_self(), SCHED_FIFO, &schedParam));

  while (1) {
    //debugCall(pthread_mutex_lock(&syncStartMut[_tid]));
    cudaDebugCall(hipStreamSynchronize(cudaStreams[_tid]));
    //debugCall(pthread_mutex_unlock(&syncEndMut[_tid]));
  }
  return NULL;
}

int main(int argc, char **argv) {
  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(0, &cpuSet);
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  // init setup
  cudaDebugCall(hipSetDeviceFlags(hipDeviceScheduleSpin));
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    cudaDebugCall(hipStreamCreate(&cudaStreams[_tid]));
    //syncStartMut[_tid] = PTHREAD_MUTEX_INITIALIZER;
    //debugCall(pthread_mutex_lock(&syncStartMut[_tid]));
    //syncEndMut[_tid] = PTHREAD_MUTEX_INITIALIZER;
    //debugCall(pthread_mutex_lock(&syncEndMut[_tid]));
  }

  pthreadDataRead();
  prioGen(atoi(argv[1]));

  fdMinGpuFreq = open("/sys/devices/17000000.gp10b/devfreq/17000000.gp10b/min_freq", O_WRONLY | O_TRUNC);
  fdMaxGpuFreq = open("/sys/devices/17000000.gp10b/devfreq/17000000.gp10b/max_freq", O_WRONLY | O_TRUNC);
  syscall(SYS_write, fdMaxGpuFreq, "1300500000", 10);
  syscall(SYS_write, fdMinGpuFreq, "1300500000", 10);
  
  // START SCHEDULING
  int _tids[PTHREAD_NUM];
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    _tids[_tid] = _tid;
    pthread_create(&mainThreads[_tid], NULL, threadFunc, (void *)&_tids[_tid]);
    cudaDebugCall(hipStreamSynchronize(cudaStreams[_tid]));
    //debugCall(pthread_create(&syncThreads[_tid], NULL, syncFunc, (void *)&_tids[_tid]));
  }
  
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    //debugCall(pthread_join(mainThreads[_tid], NULL));
    pthread_join(mainThreads[_tid], NULL);
    //pthread_join(mainThreads[_tid], NULL);
  }

  //cudaDebugCall(hipDeviceReset());
  //hipDeviceReset();

  close(fdMinGpuFreq);
  close(fdMaxGpuFreq);
  exit(timeExceeded);
  return 0;
}
