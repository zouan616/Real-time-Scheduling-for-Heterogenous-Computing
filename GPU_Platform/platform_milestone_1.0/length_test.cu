/*
USAGE:
  sudo ./length_test [taskType] [taskLen] [unitTask]
  ctrl + c to terminate
  nvprof to inspect gpu tasks test results
EFFECT:
  test the time cost of CPU/GPU tasks
ARGS:
  [taskType]: 0 -> cpu, 1 -> gpu
  [taskLen]: (float) task length of a single task, in ms
  [unitTask]: (int) number of iterations to make up a 1 ms task
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define cudaDebugCall(F)                                                                                               \
  if ((F) != hipSuccess) {                                                                                            \
    printf("Error at line %d: %s\n", __LINE__, hipGetErrorString(hipGetLastError()));                                \
    exit(1);                                                                                                           \
  };

int taskType;
float taskLen;
int cpuUnitTask;
__device__ int gpuUnitTask;
__device__ float deviceData[2048];

void cpuTaskFunc(float cpuTaskLen) {
  float c = 0;
  long i = cpuTaskLen * cpuUnitTask;
  for (long j = 0; j < i; ++j) {
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
  }
}

__global__ void gpuTaskFunc(float gpuTaskLen) {
  long i = threadIdx.x + blockIdx.x * blockDim.x;
  long j = gpuTaskLen * gpuUnitTask;
  float a = 9876.54321, b = 543.21;
  for (long k = 0; k < j; ++k) {
    deviceData[i] += a / b;
    deviceData[i] -= a / b;
  }
}

int main(int argc, char **argv) {
  taskType = atoi(argv[1]);
  taskLen = atof(argv[2]);

  if (taskType == 0) {
    cpuUnitTask = atoi(argv[3]);
    while (1) {
      struct timeval startTime;
      struct timeval endTime;
      long duration;
      gettimeofday(&startTime, NULL);
      cpuTaskFunc(taskLen);
      gettimeofday(&endTime, NULL);
      duration = endTime.tv_sec * 1000000 + endTime.tv_usec - (startTime.tv_sec * 1000000 + startTime.tv_usec);
      printf("%ld\n", duration);
    }
  } else {
    int tmp = atoi(argv[3]);
    cudaDebugCall(hipMemcpyToSymbol(HIP_SYMBOL(gpuUnitTask), &tmp, sizeof(int)));
    gpuTaskFunc<<<2, 1024, 0, 0>>>(250); // gpu warm up
    cudaDebugCall(hipDeviceSynchronize());
    while (1) {
      gpuTaskFunc<<<2, 1024, 0, 0>>>(taskLen);
      cudaDebugCall(hipDeviceSynchronize());
    }
  }

  return 0;
}
