#include "hip/hip_runtime.h"
#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <unistd.h>

__global__ void one() { printf("1\n"); }

void *call_one(void *a) {
  cpu_set_t mask;
  CPU_ZERO(&mask);
  CPU_SET(1, &mask);
  pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &mask);
  printf("Calling 1\n");
  one<<<2, 1024>>>();
  hipDeviceSynchronize();
  return NULL;
}

__global__ void two() { printf("2\n"); }

void *call_two(void *a) {
  cpu_set_t mask;
  CPU_ZERO(&mask);
  CPU_SET(1, &mask);
  pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &mask);
  printf("Calling 2\n");
  two<<<2, 1024>>>();
  hipDeviceSynchronize();
  return NULL;
}

__global__ void three() { printf("3\n"); }

void *call_three(void *a) {
  cpu_set_t mask;
  CPU_ZERO(&mask);
  CPU_SET(1, &mask);
  pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &mask);
  printf("Calling 3\n");
  three<<<2, 1024>>>();
  hipDeviceSynchronize();
  return NULL;
}

__global__ void four() { printf("4\n"); }

void *call_four(void *a) {
  cpu_set_t mask;
  CPU_ZERO(&mask);
  CPU_SET(1, &mask);
  pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &mask);
  printf("Calling 4\n");
  four<<<2, 1024>>>();
  hipDeviceSynchronize();
  return NULL;
}

int main() {
  pthread_attr_t attr;
  pthread_attr_init(&attr);

  struct sched_param param = {99};
  pthread_attr_setschedparam(&attr, &param);
  pthread_attr_setschedpolicy(&attr, SCHED_FIFO);

  pthread_t thr[4];

  pthread_create(&thr[0], &attr, call_one, &mask);
  pthread_create(&thr[1], &attr, call_two, &mask);
  pthread_create(&thr[2], &attr, call_three, &mask);
  pthread_create(&thr[3], &attr, call_four, &mask);

  for (int i = 0; i < 4; ++i)
    pthread_join(thr[i], NULL);

  return 0;
}
