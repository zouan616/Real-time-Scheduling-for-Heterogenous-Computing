#include "hip/hip_runtime.h"
// sudo ./scheduling_experiment [priorityOrder]

#include "scheduling_experiment.h"

// int fdMinGpuFreq;
// int fdMaxGpuFreq;
// unsigned long gpuAvaiFreqs[] = {
//     114750000, 216750000, 318750000,  420750000,  522750000,  624750000, 726750000,
//     854250000, 930750000, 1032750000, 1122000000, 1236750000, 1300500000}; // default = 114750000
// const char * gpuAvaiFreqs_cpy[] = {
//     "114750000", "216750000", "318750000", "420750000", "522750000", "624750000", "726750000",
//     "854250000", "930750000", "1032750000", "1122000000", "1236750000", "1300500000"};

void cpuTaskFunc(float cpuTaskLen) {
  float c = 0;
  long i = cpuTaskLen * CPU_UNIT_TASK;
  for (long j = 0; j < i; ++j) {
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
    c += 98765.4321 / 654.321;
    c -= 98765.4321 / 654.321;
  }
}

__global__ void gpuTaskFunc(int _tid, float gpuTaskLen) {
  long i = threadIdx.x + blockIdx.x * blockDim.x;
  long j = gpuTaskLen * GPU_UNIT_TASK;
  float a = 9876.54321, b = 543.21;
  for (long k = 0; k < j; ++k) {
    deviceData[_tid][i] += a / b;
    deviceData[_tid][i] -= a / b;
  }
}

void *threadFunc(void *_tidPtr) {
  int _tid = *(int *)_tidPtr;
  long ddlusec = ddls[_tid] * 1000;

  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(3, &cpuSet); // allocate to cpu no.3
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  struct sched_param schedParam;
  schedParam.sched_priority = prios[_tid];
  debugCall(pthread_setschedparam(pthread_self(), SCHED_FIFO, &schedParam));
  // syscall(SYS_write, fdMaxGpuFreq, "1300500000", 10);
  // syscall(SYS_write, fdMinGpuFreq, "1300500000", 10);

  struct timeval startTime;
  struct timeval endTime;
  long duration; // us

  struct timeval cpuSegStartTime;
  struct timeval cpuSegEndTime;
  long cpuSegDuration; // us
  long diffDuration;   // us
  float gpuRatio;
  float targetGpuRatio;

  int extension = 0;

  // MAIN LOOP
  for (int i = 0; i < 100; ++i) {
    gettimeofday(&startTime, NULL);
    for (int j = 0; j < cpuTaskNum[_tid] - 1; ++j) {
      cpuTaskFunc(cpuTaskLens[_tid][j]);
      gpuTaskFunc<<<2, 1024, 0, cudaStreams[_tid]>>>(_tid, gpuTaskLens[_tid][j]);
    }
    cpuTaskFunc(cpuTaskLens[_tid][cpuTaskNum[_tid] - 1]);
    gettimeofday(&endTime, NULL);

    // some other pthreads time exceeded
    if (timeExceeded) {
      return NULL;
    }
    // current pthread time exceeded
    duration = endTime.tv_sec * 1000000 + endTime.tv_usec - (startTime.tv_sec * 1000000 + startTime.tv_usec);
    if (duration > ddlusec) {
      timeExceeded = 1;
      return NULL;
    }
    // current pthread not time exceeded, sleep until deadline
    usleep(ddlusec - duration);
  }
  // current pthread successfully scheduled
  return NULL;
}

void *syncFunc(void *_tidPtr) {
  int _tid = *(int *)_tidPtr;

  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(0, &cpuSet);
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  struct sched_param schedParam;
  schedParam.sched_priority = 99;
  debugCall(pthread_setschedparam(pthread_self(), SCHED_FIFO, &schedParam));

  while (1) {
    cudaDebugCall(hipStreamSynchronize(cudaStreams[_tid]));
  }
  return NULL;
}

int main(int argc, char **argv) {
  cpu_set_t cpuSet;
  CPU_ZERO(&cpuSet);
  CPU_SET(0, &cpuSet);
  debugCall(pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuSet));

  // init setup
  cudaDebugCall(hipSetDeviceFlags(hipDeviceScheduleSpin));
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    cudaDebugCall(hipStreamCreate(&cudaStreams[_tid]));
  }

  pthreadDataRead();
  prioGen(atoi(argv[1]));

  // fdMinGpuFreq = open("/sys/devices/17000000.gp10b/devfreq/17000000.gp10b/min_freq", O_WRONLY | O_TRUNC);
  // fdMaxGpuFreq = open("/sys/devices/17000000.gp10b/devfreq/17000000.gp10b/max_freq", O_WRONLY | O_TRUNC);

  //gpu warmup
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    gpuTaskFunc<<<2, 1024, 0, cudaStreams[_tid]>>>(_tid, 250);
  }
  usleep(250000);
  cudaDebugCall(hipDeviceSynchronize());

  printf("start scheduling\n");

  // START SCHEDULING
  int _tids[PTHREAD_NUM];
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    _tids[_tid] = _tid;
    debugCall(pthread_create(&mainThreads[_tid], NULL, threadFunc, (void *)&_tids[_tid]));
    debugCall(pthread_create(&syncThreads[_tid], NULL, syncFunc, (void *)&_tids[_tid]));
  }
  // Thread join
  for (int _tid = 0; _tid < PTHREAD_NUM; ++_tid) {
    debugCall(pthread_join(mainThreads[_tid], NULL));
  }
  cudaDebugCall(hipDeviceReset());

  // close(fdMinGpuFreq);
  // close(fdMaxGpuFreq);
  exit(timeExceeded);
  return 0;
}
